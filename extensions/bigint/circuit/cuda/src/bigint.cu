#include "hip/hip_runtime.h"
#include "launcher.cuh"
#include "primitives/buffer_view.cuh"
#include "primitives/histogram.cuh"
#include "primitives/trace_access.h"
#include "rv32-adapters/heap.cuh"
#include "rv32-adapters/heap_branch.cuh"
#include "rv32im/cores/alu.cuh"
#include "rv32im/cores/beq.cuh"
#include "rv32im/cores/blt.cuh"
#include "rv32im/cores/less_than.cuh"
#include "rv32im/cores/mul.cuh"
#include "rv32im/cores/shift.cuh"

using namespace riscv;

constexpr size_t INT256_NUM_LIMBS = 32;

using BaseAlu256CoreRecord = BaseAluCoreRecord<32>;
using BaseAlu256Core = BaseAluCore<32>;
template <typename T> using BaseAlu256CoreCols = BaseAluCoreCols<T, 32>;

using BranchEqual256Core = BranchEqualCore<32>;
template <typename T> using BranchEqual256CoreCols = BranchEqualCoreCols<T, 32>;
using BranchEqual256CoreRecord = BranchEqualCoreRecord<32>;

using LessThan256CoreRecord = LessThanCoreRecord<32>;
using LessThan256Core = LessThanCore<32>;
template <typename T> using LessThan256CoreCols = LessThanCoreCols<T, 32>;

using Multiplication256CoreRecord = MultiplicationCoreRecord<32>;
using Multiplication256Core = MultiplicationCore<32>;
template <typename T> using Multiplication256CoreCols = MultiplicationCoreCols<T, 32>;

using Shift256CoreRecord = ShiftCoreRecord<32>;
using Shift256Core = ShiftCore<32>;
template <typename T> using Shift256CoreCols = ShiftCoreCols<T, 32>;

using BranchLessThan256CoreRecord = BranchLessThanCoreRecord<32>;
using BranchLessThan256Core = BranchLessThanCore<32>;
template <typename T> using BranchLessThan256CoreCols = BranchLessThanCoreCols<T, 32>;

// Heap adapter instantiation for 256-bit operations
// NUM_READS = 2, READ_SIZE = INT256_NUM_LIMBS (32 bytes), WRITE_SIZE = INT256_NUM_LIMBS (32 bytes)
// BLOCKS_PER_READ = 1, BLOCKS_PER_WRITE = 1
using Rv32HeapAdapterExecutor256 = Rv32HeapAdapterExecutor<2, INT256_NUM_LIMBS, INT256_NUM_LIMBS>;

template <typename T> struct BaseAlu256Cols {
    Rv32HeapAdapterCols<T, 2, INT256_NUM_LIMBS, INT256_NUM_LIMBS> adapter;
    BaseAlu256CoreCols<T> core;
};

struct BaseAlu256Record {
    Rv32HeapAdapterRecord<2, INT256_NUM_LIMBS, INT256_NUM_LIMBS> adapter;
    BaseAlu256CoreRecord core;
};

__global__ void alu256_tracegen(
    Fp *d_trace,
    size_t height,
    DeviceBufferConstView<BaseAlu256Record> d_records,
    uint32_t *d_range_checker_ptr,
    size_t range_checker_bins,
    uint32_t *d_bitwise_lookup_ptr,
    size_t bitwise_num_bits,
    uint32_t pointer_max_bits,
    uint32_t timestamp_max_bits
) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    RowSlice row(d_trace + idx, height);
    if (idx < d_records.len()) {
        auto const &rec = d_records[idx];

        Rv32HeapAdapterExecutor256 adapter(
            pointer_max_bits,
            VariableRangeChecker(d_range_checker_ptr, range_checker_bins),
            BitwiseOperationLookup(d_bitwise_lookup_ptr, bitwise_num_bits),
            timestamp_max_bits
        );
        adapter.fill_trace_row(row, rec.adapter);

        BaseAlu256Core core(BitwiseOperationLookup(d_bitwise_lookup_ptr, bitwise_num_bits));
        core.fill_trace_row(row.slice_from(COL_INDEX(BaseAlu256Cols, core)), rec.core);
    } else {
        row.fill_zero(0, sizeof(BaseAlu256Cols<uint8_t>));
    }
}

extern "C" int _alu256_tracegen(
    Fp *d_trace,
    size_t height,
    size_t width,
    DeviceBufferConstView<BaseAlu256Record> d_records,
    uint32_t *d_range_checker_ptr,
    size_t range_checker_bins,
    uint32_t *d_bitwise_lookup_ptr,
    size_t bitwise_num_bits,
    uint32_t pointer_max_bits,
    uint32_t timestamp_max_bits
) {
    assert((height & (height - 1)) == 0);
    assert(height >= d_records.len());
    assert(width == sizeof(BaseAlu256Cols<uint8_t>));

    auto [grid, block] = kernel_launch_params(height, 256);
    alu256_tracegen<<<grid, block>>>(
        d_trace,
        height,
        d_records,
        d_range_checker_ptr,
        range_checker_bins,
        d_bitwise_lookup_ptr,
        bitwise_num_bits,
        pointer_max_bits,
        timestamp_max_bits
    );
    return CHECK_KERNEL();
}

// Heap branch adapter instantiation for 256-bit operations
// NUM_READS = 2, READ_SIZE = INT256_NUM_LIMBS (32 bytes)
using Rv32HeapBranchAdapter256 = Rv32HeapBranchAdapter<2, INT256_NUM_LIMBS>;

template <typename T> struct BranchEqual256Cols {
    Rv32HeapBranchAdapterCols<T, 2, INT256_NUM_LIMBS> adapter;
    BranchEqual256CoreCols<T> core;
};

struct BranchEqual256Record {
    Rv32HeapBranchAdapterRecord<2> adapter;
    BranchEqual256CoreRecord core;
};

__global__ void branch_equal256_tracegen(
    Fp *d_trace,
    size_t height,
    DeviceBufferConstView<BranchEqual256Record> d_records,
    uint32_t *d_range_checker_ptr,
    size_t range_checker_bins,
    uint32_t *d_bitwise_lookup_ptr,
    size_t bitwise_num_bits,
    uint32_t pointer_max_bits,
    uint32_t timestamp_max_bits
) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    RowSlice row(d_trace + idx, height);
    if (idx < d_records.len()) {
        auto const &rec = d_records[idx];

        Rv32HeapBranchAdapter256 adapter(
            pointer_max_bits,
            VariableRangeChecker(d_range_checker_ptr, range_checker_bins),
            BitwiseOperationLookup(d_bitwise_lookup_ptr, bitwise_num_bits),
            timestamp_max_bits
        );
        adapter.fill_trace_row(row, rec.adapter);

        BranchEqual256Core core;
        core.fill_trace_row(row.slice_from(COL_INDEX(BranchEqual256Cols, core)), rec.core);
    } else {
        row.fill_zero(0, sizeof(BranchEqual256Cols<uint8_t>));
    }
}

extern "C" int _branch_equal256_tracegen(
    Fp *d_trace,
    size_t height,
    size_t width,
    DeviceBufferConstView<BranchEqual256Record> d_records,
    uint32_t *d_range_checker_ptr,
    size_t range_checker_bins,
    uint32_t *d_bitwise_lookup_ptr,
    size_t bitwise_num_bits,
    uint32_t pointer_max_bits,
    uint32_t timestamp_max_bits
) {
    assert((height & (height - 1)) == 0);
    assert(height >= d_records.len());
    assert(width == sizeof(BranchEqual256Cols<uint8_t>));

    auto [grid, block] = kernel_launch_params(height, 256);
    branch_equal256_tracegen<<<grid, block>>>(
        d_trace,
        height,
        d_records,
        d_range_checker_ptr,
        range_checker_bins,
        d_bitwise_lookup_ptr,
        bitwise_num_bits,
        pointer_max_bits,
        timestamp_max_bits
    );
    return CHECK_KERNEL();
}

template <typename T> struct LessThan256Cols {
    Rv32HeapAdapterCols<T, 2, INT256_NUM_LIMBS, INT256_NUM_LIMBS> adapter;
    LessThan256CoreCols<T> core;
};

struct LessThan256Record {
    Rv32HeapAdapterRecord<2, INT256_NUM_LIMBS, INT256_NUM_LIMBS> adapter;
    LessThan256CoreRecord core;
};

__global__ void less_than256_tracegen(
    Fp *d_trace,
    size_t height,
    DeviceBufferConstView<LessThan256Record> d_records,
    uint32_t *d_range_checker_ptr,
    size_t range_checker_bins,
    uint32_t *d_bitwise_lookup_ptr,
    size_t bitwise_num_bits,
    uint32_t pointer_max_bits,
    uint32_t timestamp_max_bits
) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    RowSlice row(d_trace + idx, height);
    if (idx < d_records.len()) {
        auto const &rec = d_records[idx];

        Rv32HeapAdapterExecutor<2, INT256_NUM_LIMBS, INT256_NUM_LIMBS> adapter(
            pointer_max_bits,
            VariableRangeChecker(d_range_checker_ptr, range_checker_bins),
            BitwiseOperationLookup(d_bitwise_lookup_ptr, bitwise_num_bits),
            timestamp_max_bits
        );
        adapter.fill_trace_row(row, rec.adapter);

        LessThan256Core core(BitwiseOperationLookup(d_bitwise_lookup_ptr, bitwise_num_bits));
        core.fill_trace_row(row.slice_from(COL_INDEX(LessThan256Cols, core)), rec.core);
    } else {
        row.fill_zero(0, sizeof(LessThan256Cols<uint8_t>));
    }
}

extern "C" int _less_than256_tracegen(
    Fp *d_trace,
    size_t height,
    size_t width,
    DeviceBufferConstView<LessThan256Record> d_records,
    uint32_t *d_range_checker_ptr,
    size_t range_checker_bins,
    uint32_t *d_bitwise_lookup_ptr,
    size_t bitwise_num_bits,
    uint32_t pointer_max_bits,
    uint32_t timestamp_max_bits
) {
    assert((height & (height - 1)) == 0);
    assert(height >= d_records.len());
    assert(width == sizeof(LessThan256Cols<uint8_t>));

    auto [grid, block] = kernel_launch_params(height, 256);
    less_than256_tracegen<<<grid, block>>>(
        d_trace,
        height,
        d_records,
        d_range_checker_ptr,
        range_checker_bins,
        d_bitwise_lookup_ptr,
        bitwise_num_bits,
        pointer_max_bits,
        timestamp_max_bits
    );
    return CHECK_KERNEL();
}

template <typename T> struct BranchLessThan256Cols {
    Rv32HeapBranchAdapterCols<T, 2, INT256_NUM_LIMBS> adapter;
    BranchLessThan256CoreCols<T> core;
};

struct BranchLessThan256Record {
    Rv32HeapBranchAdapterRecord<2> adapter;
    BranchLessThan256CoreRecord core;
};

__global__ void branch_less_than256_tracegen(
    Fp *d_trace,
    size_t height,
    DeviceBufferConstView<BranchLessThan256Record> d_records,
    uint32_t *d_range_checker_ptr,
    size_t range_checker_bins,
    uint32_t *d_bitwise_lookup_ptr,
    size_t bitwise_num_bits,
    uint32_t pointer_max_bits,
    uint32_t timestamp_max_bits
) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    RowSlice row(d_trace + idx, height);
    if (idx < d_records.len()) {
        auto const &rec = d_records[idx];

        Rv32HeapBranchAdapter256 adapter(
            pointer_max_bits,
            VariableRangeChecker(d_range_checker_ptr, range_checker_bins),
            BitwiseOperationLookup(d_bitwise_lookup_ptr, bitwise_num_bits),
            timestamp_max_bits
        );
        adapter.fill_trace_row(row, rec.adapter);

        BranchLessThan256Core core(BitwiseOperationLookup(d_bitwise_lookup_ptr, bitwise_num_bits));
        core.fill_trace_row(row.slice_from(COL_INDEX(BranchLessThan256Cols, core)), rec.core);
    } else {
        row.fill_zero(0, sizeof(BranchLessThan256Cols<uint8_t>));
    }
}

extern "C" int _branch_less_than256_tracegen(
    Fp *d_trace,
    size_t height,
    size_t width,
    DeviceBufferConstView<BranchLessThan256Record> d_records,
    uint32_t *d_range_checker_ptr,
    size_t range_checker_bins,
    uint32_t *d_bitwise_lookup_ptr,
    size_t bitwise_num_bits,
    uint32_t pointer_max_bits,
    uint32_t timestamp_max_bits
) {
    assert((height & (height - 1)) == 0);
    assert(height >= d_records.len());
    assert(width == sizeof(BranchLessThan256Cols<uint8_t>));

    auto [grid, block] = kernel_launch_params(height, 256);
    branch_less_than256_tracegen<<<grid, block>>>(
        d_trace,
        height,
        d_records,
        d_range_checker_ptr,
        range_checker_bins,
        d_bitwise_lookup_ptr,
        bitwise_num_bits,
        pointer_max_bits,
        timestamp_max_bits
    );
    return CHECK_KERNEL();
}

template <typename T> struct Shift256Cols {
    Rv32HeapAdapterCols<T, 2, INT256_NUM_LIMBS, INT256_NUM_LIMBS> adapter;
    Shift256CoreCols<T> core;
};

struct Shift256Record {
    Rv32HeapAdapterRecord<2, INT256_NUM_LIMBS, INT256_NUM_LIMBS> adapter;
    Shift256CoreRecord core;
};

__global__ void shift256_tracegen(
    Fp *d_trace,
    size_t height,
    DeviceBufferConstView<Shift256Record> d_records,
    uint32_t *d_range_checker_ptr,
    size_t range_checker_bins,
    uint32_t *d_bitwise_lookup_ptr,
    size_t bitwise_num_bits,
    uint32_t pointer_max_bits,
    uint32_t timestamp_max_bits
) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    RowSlice row(d_trace + idx, height);
    if (idx < d_records.len()) {
        auto const &rec = d_records[idx];

        Rv32HeapAdapterExecutor<2, INT256_NUM_LIMBS, INT256_NUM_LIMBS> adapter(
            pointer_max_bits,
            VariableRangeChecker(d_range_checker_ptr, range_checker_bins),
            BitwiseOperationLookup(d_bitwise_lookup_ptr, bitwise_num_bits),
            timestamp_max_bits
        );
        adapter.fill_trace_row(row, rec.adapter);

        Shift256Core core(
            BitwiseOperationLookup(d_bitwise_lookup_ptr, bitwise_num_bits),
            VariableRangeChecker(d_range_checker_ptr, range_checker_bins)
        );
        core.fill_trace_row(row.slice_from(COL_INDEX(Shift256Cols, core)), rec.core);
    } else {
        row.fill_zero(0, sizeof(Shift256Cols<uint8_t>));
    }
}

extern "C" int _shift256_tracegen(
    Fp *d_trace,
    size_t height,
    size_t width,
    DeviceBufferConstView<Shift256Record> d_records,
    uint32_t *d_range_checker_ptr,
    size_t range_checker_bins,
    uint32_t *d_bitwise_lookup_ptr,
    size_t bitwise_num_bits,
    uint32_t pointer_max_bits,
    uint32_t timestamp_max_bits
) {
    assert((height & (height - 1)) == 0);
    assert(height >= d_records.len());
    assert(width == sizeof(Shift256Cols<uint8_t>));

    auto [grid, block] = kernel_launch_params(height, 256);
    shift256_tracegen<<<grid, block>>>(
        d_trace,
        height,
        d_records,
        d_range_checker_ptr,
        range_checker_bins,
        d_bitwise_lookup_ptr,
        bitwise_num_bits,
        pointer_max_bits,
        timestamp_max_bits
    );
    return CHECK_KERNEL();
}

template <typename T> struct Multiplication256Cols {
    Rv32HeapAdapterCols<T, 2, INT256_NUM_LIMBS, INT256_NUM_LIMBS> adapter;
    Multiplication256CoreCols<T> core;
};

struct Multiplication256Record {
    Rv32HeapAdapterRecord<2, INT256_NUM_LIMBS, INT256_NUM_LIMBS> adapter;
    Multiplication256CoreRecord core;
};

__global__ void multiplication256_tracegen(
    Fp *d_trace,
    size_t height,
    DeviceBufferConstView<Multiplication256Record> d_records,
    uint32_t *d_range_checker_ptr,
    size_t range_checker_bins,
    uint32_t *d_bitwise_lookup_ptr,
    size_t bitwise_num_bits,
    uint32_t *d_range_tuple_ptr,
    uint2 range_tuple_sizes,
    uint32_t pointer_max_bits,
    uint32_t timestamp_max_bits
) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    RowSlice row(d_trace + idx, height);
    if (idx < d_records.len()) {
        auto const &rec = d_records[idx];

        Rv32HeapAdapterExecutor<2, INT256_NUM_LIMBS, INT256_NUM_LIMBS> adapter(
            pointer_max_bits,
            VariableRangeChecker(d_range_checker_ptr, range_checker_bins),
            BitwiseOperationLookup(d_bitwise_lookup_ptr, bitwise_num_bits),
            timestamp_max_bits
        );
        adapter.fill_trace_row(row, rec.adapter);

        RangeTupleChecker<2> range_tuple_checker(
            d_range_tuple_ptr, (uint32_t[2]){range_tuple_sizes.x, range_tuple_sizes.y}
        );
        Multiplication256Core core(range_tuple_checker);
        core.fill_trace_row(row.slice_from(COL_INDEX(Multiplication256Cols, core)), rec.core);
    } else {
        row.fill_zero(0, sizeof(Multiplication256Cols<uint8_t>));
    }
}

extern "C" int _multiplication256_tracegen(
    Fp *d_trace,
    size_t height,
    size_t width,
    DeviceBufferConstView<Multiplication256Record> d_records,
    uint32_t *d_range_checker_ptr,
    size_t range_checker_bins,
    uint32_t *d_bitwise_lookup_ptr,
    size_t bitwise_num_bits,
    uint32_t *d_range_tuple_ptr,
    uint2 range_tuple_sizes,
    uint32_t pointer_max_bits,
    uint32_t timestamp_max_bits
) {
    assert((height & (height - 1)) == 0);
    assert(height >= d_records.len());
    assert(width == sizeof(Multiplication256Cols<uint8_t>));

    auto [grid, block] = kernel_launch_params(height, 256);
    multiplication256_tracegen<<<grid, block>>>(
        d_trace,
        height,
        d_records,
        d_range_checker_ptr,
        range_checker_bins,
        d_bitwise_lookup_ptr,
        bitwise_num_bits,
        d_range_tuple_ptr,
        range_tuple_sizes,
        pointer_max_bits,
        timestamp_max_bits
    );
    return CHECK_KERNEL();
}
